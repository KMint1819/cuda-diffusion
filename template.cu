#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "helper.hpp"


#define TILE_SIZE 32

__global__ void testKernel(float* originalVal, float* newVal, int valSize) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= 0 && i < valSize)
    newVal[i] = 91.035789; //originalVal[i] + 1;
}

void launchTest(float* originalVal, float* newVal, int valSize) {

  // INSERT CODE HERE
  int X_size = ceil(1.0 * valSize / TILE_SIZE);
  dim3 DimGrid(X_size,1,1);
  dim3 DimBlock(TILE_SIZE,1,1);
  testKernel<<<DimGrid,DimBlock>>>(originalVal,newVal,valSize);
}

static int eval(const int nx) {

  // Generate model
  const auto conf_info = std::string("test[") + std::to_string(nx) + "]";
  INFO("Running "  << conf_info);

  // generate input data
  timer_start("Generating test data");
  std::vector<float> hostA0(nx);
  // generate_data(hostA0.data(), nx, 1, 1);
  read_input_data(hostA0.data(), nx);
  std::vector<float> hostAnext(nx);

  timer_start("Allocating GPU memory.");
  float *deviceA0 = nullptr, *deviceAnext = nullptr;
  CUDA_RUNTIME(hipMalloc((void **)&deviceA0, nx * sizeof(float)));
  CUDA_RUNTIME(hipMalloc((void **)&deviceAnext, nx * sizeof(float)));
  timer_stop();

  timer_start("Copying inputs to the GPU.");
  CUDA_RUNTIME(hipMemcpy(deviceA0, hostA0.data(), nx * sizeof(float), hipMemcpyDefault));
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  timer_start("Performing GPU convlayer");
  launchTest(deviceA0, deviceAnext, nx);
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  timer_start("Copying output to the CPU");
  CUDA_RUNTIME(hipMemcpy(hostAnext.data(), deviceAnext, nx * sizeof(float), hipMemcpyDefault));
  CUDA_RUNTIME(hipDeviceSynchronize());
  timer_stop();

  // verify with provided implementation
  timer_start("Verifying results");
  verify(hostAnext.data(), nx, 1, 1);
  timer_stop();

  CUDA_RUNTIME(hipFree(deviceA0));
  CUDA_RUNTIME(hipFree(deviceAnext));

  return 0;
}

TEST_CASE("Group 10", "[gten]") {

  SECTION("32") {
    eval(32);
  }

  // SECTION("1024") {
  //   eval(1024);
  // }

  // SECTION("2048") {
  //   eval(2048);
  // }
}
